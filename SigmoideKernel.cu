
#include <hip/hip_runtime.h>
﻿extern "C"
__global__ void SigmoideKernel(float* __restrict__ a, float* __restrict__ c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = 1.0f / (1.0f + expf(-a[idx]));
    }
}